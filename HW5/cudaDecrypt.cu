#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.h"


void readKeyInfo(char* fileName, unsigned int* n, unsigned int* p, unsigned int* g, unsigned int* h);
void readMessage(char* fileName, unsigned int* cipherText, unsigned int* a);

__device__ int kernel_modprod(unsigned int a, unsigned int b, unsigned int p) {
    unsigned int za = a;
    unsigned int ab = 0;

    while(b > 0) {
        if (b % 2 == 1) ab = (ab + za) % p;
        za = (2 * za) % p;
        b /= 2;
    }
    return ab;
}

__device__ int kernel_modExp(unsigned int a, unsigned int b, unsigned int p) {
    unsigned int z = a;
    unsigned int aExpb = 1;

    while (b > 0) {
        if (b % 2 == 1) aExp = kernel_modprod(aExpb, z, p);
        z = modprod(z, z, p);
        b /= 2;
    }
    return aExpb;
}

__global__ int findSecretKey(unsigned int g, unsigned int p, unsigned int h,  unsigned int* h_x) {
  
  // find the secret key
  int tId = threadIdx.x;
  int bId = blockIdx.x;
  int bSize = blockDim.x;

  unsigned int x = tId + bId*bSize;

  //recreate modProd and ModExp
  if (kernel_modExp(g,x,p) == h) {
    hipMemcpy(x, h_x, sizeof(unsigned int),hipMemcpyDeviceToHost);
  }

  //create mem for x to copy from device to host
}

int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  
  readKeyInfo("public_key.txt", &n, &p, &g, &h);

  printf("Decrypt.c Read in:\nn = %u\np = %u\ng = %u\nh = %u\n",
          n, p, g, h);
    
  //copy values to device
  hipMemcpy(n,d_n,sizeof(unsigned int),hipMemcpyHostToDevice);
  hipMemcpy(p,d_p,sizeof(unsigned int),hipMemcpyHostToDevice);
  hipMemcpy(g,d_g,sizeof(unsigned int),hipMemcpyHostToDevice);
  hipMemcpy(h,d_h,sizeof(unsigned int),hipMemcpyHostToDevice);

  FILE* file;
  file = fopen("message.txt", "r");
  fscanf(file, "%u", &Nints);
  fclose(file);

  unsigned int* cipherText; 
  hipMalloc(&cipherText, Nints*sizeof(unsigned int));
  
  hipMemcpy(cipherText, h_cipherText, Nints*sizeof(unsigned int), hipMemcpyDeviceToHost);

  unsigned int* a;
  hipMalloc(&a, Nints*sizeof(unsigned int));

  hipMemcpy(a, h_a, Nints*sizeof(unsigned int), hipMemcpyDeviceToHost);
  
  readMessage("message.txt", cipherText, a);

  for (int i = 0; i < Nints; i++) {
    printf("cT[%d] = %u\n", i, cipherText[i]);
  }

    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }

  /* Q3 After finding the secret key, decrypt the message */
  
  unsigned int x;
  findSecretKey << x >> (g, p, h);
  hipDeviceSynchronize();

  ElGamalDecrypt(cipherText, a, Nints, p, x);

  char *plainText;
    
  unsigned int Nchars = Nints*(n-1)/8;

  convertZToString(cipherText, Nints, plainText, Nchars);

  printf("Decrypted Message: %s\n", plainText);

  hipFree(cipherText);
  hipFree(a);

  return 0;
}

void readKeyInfo(char* fileName, unsigned int* n, unsigned int* p, unsigned int* g, unsigned int* h) {
    
    FILE *file;
    file = fopen(fileName, "r");

    char *buff;
     
    for (int i = 0; i < 4; i++) {
        if (i == 0) fscanf(file, "%u", n);
        if (i == 1) fscanf(file, "%u", p);
        if (i == 2) fscanf(file, "%u", g);
        if (i == 3) fscanf(file, "%u", h);
    }

    fclose(file); 

}

void readMessage(char* fileName, unsigned int* m, unsigned int* a) {

    FILE *file;
    file = fopen(fileName, "r"); //open the file with writing privaledges
  
    unsigned int Nints;
    fscanf(file, "%u", &Nints);

    for (int i = 0; i < Nints; i++) {
        fscanf(file, "%u", &m[i]);
        fscanf(file, "%u", &a[i]); 
    }

    //close the file when done writing
    fclose(file);

}

nt findSecretKey() {
  
  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }

}
