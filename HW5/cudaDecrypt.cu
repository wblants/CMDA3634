#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

//----Function Stubs-----
void readKeyInfo(const char* fileName, unsigned int* n, unsigned int* p, unsigned int* g, unsigned int* h);

void readMessage(const char* fileName, unsigned int* cipherText, unsigned int* a);

//----kernel and device functions----
__device__ int kernel_modprod(unsigned int a, unsigned int b, unsigned int p) {
    unsigned int za = a;
    unsigned int ab = 0;

    while(b > 0) {
        if (b % 2 == 1) ab = (ab + za) % p;
        za = (2 * za) % p;
        b /= 2;
    }
    return ab;
}

__device__ int kernel_modExp(unsigned int a, unsigned int b, unsigned int p) {
    unsigned int z = a;
    unsigned int aExpb = 1;

    while (b > 0) {
        if (b % 2 == 1) aExpb = kernel_modprod(aExpb, z, p);
        z = kernel_modprod(z, z, p);
        b /= 2;
    }
    return aExpb;
}

__global__ void findSecretKey(unsigned int g, unsigned int p, unsigned int h, unsigned int* d_x) {
  
  // find the secret key
  int tId = threadIdx.x;
  int bId = blockIdx.x;
  int bSize = blockDim.x;

  unsigned int x = tId + bId*bSize;
  
  if (kernel_modExp(g,x,p) == h) {
      *d_x = x;
  }

}

int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;
  unsigned int Nthreads = atoi(argv[1]);

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  
  readKeyInfo("bonus_public_key.txt", &n, &p, &g, &h);

  printf("Decrypt.c Read in:\nn = %u\np = %u\ng = %u\nh = %u\n",
          n, p, g, h);
  
  //block dimmensions
  dim3 B(Nthreads, 1, 1);

  //grid dimmensions
  dim3 G((p+Nthreads-1)/Nthreads, 1, 1);
 
  FILE* file;
  file = fopen("bonus_message.txt", "r");
  fscanf(file, "%u", &Nints);
  fclose(file);

  unsigned int* cipherText = 
      (unsigned int*) malloc(Nints*sizeof(unsigned int));
  
  unsigned int* a =
      (unsigned int*) malloc(Nints*sizeof(unsigned int));

  //readMessage("message.txt", cipherText, a);
  readMessage("bonus_message.txt", cipherText, a);

    double startTime = clock();
      
    unsigned int *d_x;
    hipMalloc(&d_x, sizeof(unsigned int));

    findSecretKey <<< G,B >>> (g, p, h, d_x);
    hipDeviceSynchronize();
    hipMemcpy(&x, d_x, sizeof(unsigned int),hipMemcpyDeviceToHost);
    
    printf("Host has x = %u\n", x);

    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);

  /* Q3 After finding the secret key, decrypt the message */
  
  ElGamalDecrypt(cipherText, a, Nints, p, x);

  unsigned int charsPerInt = Nints*(n-1)/8;
    
  unsigned char *plainText = 
    (unsigned char*) malloc(Nints*charsPerInt*sizeof(char));

  convertZToString(cipherText, Nints, plainText, charsPerInt);

  printf("Decrypted Message: %s\n", plainText);

  return 0;
}

void readKeyInfo(const char* fileName, unsigned int* n, unsigned int* p, unsigned int* g, unsigned int* h) {
    
    FILE *file;
    file = fopen(fileName, "r");

    for (int i = 0; i < 4; i++) {
        if (i == 0) fscanf(file, "%u", n);
        if (i == 1) fscanf(file, "%u", p);
        if (i == 2) fscanf(file, "%u", g);
        if (i == 3) fscanf(file, "%u", h);
    }

    fclose(file); 

}

void readMessage(const char* fileName, unsigned int* m, unsigned int* a) {

    FILE *file;
    file = fopen(fileName, "r"); //open the file with writing privaledges
  
    unsigned int Nints;
    fscanf(file, "%u", &Nints);

    for (int i = 0; i < Nints; i++) {
        fscanf(file, "%u", &m[i]);
        fscanf(file, "%u", &a[i]); 
    }

    //close the file when done writing
    fclose(file);

}

